#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 10000
__global__ void add(int **a, int **b, int **c) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    c[x*gridDim.x] = a[x][y] + b[x][y];
}
int main(void) {
    int a[N][N], b[N][N], c[N][N];
    int **dev_a, **dev_b, **dev_c;

    hipMalloc( (void ***) &dev_a, N*N*sizeof(int));
    hipMalloc( (void ***) &dev_b, N*N*sizeof(int));
    hipMalloc( (void ***) &dev_c, N*N*sizeof(int));
    
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i][j] = i*j;
            b[i][j] = i*j;
        }
    }

    hipMemcpy( dev_a, a, N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( dev_b, b, N*N*sizeof(int), hipMemcpyHostToDevice);
    
    dim3 grid(N,N);
    add<<<grid,1>>>(dev_a,dev_b,dev_c);
    
    hipMemcpy(c, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf ("%d + %d = %d\n", a[i][j], b[i][j], c[i][j]);
        }
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
