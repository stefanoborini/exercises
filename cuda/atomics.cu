int main(void) {
    unsigned char *dev_buffer;
    unsigned int *dev_histogram;

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMalloc( (void **)&dev_buffer, 1024*sizeof(unsigned char));
    hipMemcpy( dev_buffer, buffer, 1024*sizeof(unsigned char), hipMemcpyHostToDevice);

    hipMalloc( (void **)&dev_histogram, 256*sizeof(unsigned int));
    hipMemset( dev_histogram, 0, 256*sizeof(unsigned int));
    kernel<<<>>>()

