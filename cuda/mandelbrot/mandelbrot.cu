
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__device__ int mandelbrot_point(float x, float y) {

    int max_iteration = 1000;
    int iteration;
    float a,b, new_a, new_b;
    a = 0.0f;
    b = 0.0f;
    new_a =0.0f;
    new_b =0.0f;
    iteration = 0;
    while (a*a + b*b <= 3.0f && iteration < max_iteration) {
        new_a = a*a - b*b + x;
        new_b = 2.0f*a*b + y;
        a = new_a;
        b = new_b;
        iteration++;
    }
    
    return (iteration == max_iteration ? 255 : iteration*10 % 255);
}

__global__ void compute_mandelbrot(int *pixels) {
    float x_center = -1.0f;
    float y_center =  0.0f;
    float x,y;
    int value;

    x = (float)x_center + 4.0f * (float)((float)blockIdx.x-(float)gridDim.x/2.0f) /(float)gridDim.x;
    y = (float)y_center + 4.0f * (float)((float)blockIdx.y-(float)gridDim.y/2.0f) /(float)gridDim.y;

    value = mandelbrot_point(x,y);

    pixels[blockIdx.x*3+blockIdx.y*gridDim.x*3] = value;
    pixels[blockIdx.x*3+blockIdx.y*gridDim.x*3+1] = value;
    pixels[blockIdx.x*3+blockIdx.y*gridDim.x*3+2] = value;
}

void print_mandelbrot(int *pixels, int width, int height ) {
    int i,j;
    for (i = 0; i < width; i++) {
        for (j = 0; j < height; j++) {
            if (pixels[i*3+width*j*3] == 255) printf("*");
            else printf(" ");
        }
        printf("|\n");
    }
}


int main(void) {
    int *pixels_dev;
    int *pixels;

    int width = 100;
    int height = 100;
    hipEvent_t start, stop;

    dim3 grid(width, height);
    float elapsed;

    hipMalloc((void **)&pixels_dev, sizeof(int) * width * height * 3);
    pixels = (int *)malloc(width*height*3*sizeof(int));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    compute_mandelbrot<<<grid,1>>>(pixels_dev);
    hipMemcpy(pixels, pixels_dev, width * height* 3 * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    

    hipEventElapsedTime(&elapsed, start, stop);
    printf("time: %f\n", elapsed);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    print_mandelbrot(pixels, width, height);
}


