
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void add(int *a, int * b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    int i;

    printf("%d\n", hipMalloc( (void **)&dev_a, sizeof(int)*N));
    hipMalloc( (void **)&dev_b, sizeof(int)*N);
    hipMalloc( (void **)&dev_c, sizeof(int)*N);
 
    for (i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2*i;
    } 
    
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N,1>>>(dev_a, dev_b, dev_c);
    
    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
  
    for (i = 0; i < N; i++) {
        printf("%d %d %d\n", a[i],b[i],c[i]);
    } 
     
    hipFree((void *)dev_a);
    hipFree((void *)dev_b);
    hipFree((void *)dev_c);

}
