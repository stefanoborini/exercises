#include <stdio.h>
int main(void) {
    hipDeviceProp_t prop;
    int count, i;

    hipGetDeviceCount(&count);
    
    for (i = 0; i < count; i++) {
        hipGetDeviceProperties(&prop, i);
        printf("%s\n", prop.name);
        printf("%d\n", prop.canMapHostMemory);
        printf("%d\n", prop.deviceOverlap);
        printf("%d\n", prop.multiProcessorCount);
        printf("%d\n", prop.integrated);
        printf("%d\n", prop.clockRate);
    }
}
